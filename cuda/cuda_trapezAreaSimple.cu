#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

__host__ __device__ double f(double x){
  return x*x;
}

__device__ double atomicAddDouble(double* address, double val) {
  unsigned long long int* address_as_ull = (unsigned long long int*)address;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));
  } while (assumed != old);
  return __longlong_as_double(old);
}

__global__ void integrate(double a, double b, int n, double h, double* result) {
  int i;
  double x, sum = 0.0;
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (i = idx + 1; i < n; i += stride) {
    x = a + i * h;
    sum += f(x);
  }

  sum *= 2.0;

  atomicAddDouble(result, sum);
}

double getCurrentTime() {
  struct timeval tv;
  gettimeofday(&tv, NULL);
  return tv.tv_sec + tv.tv_usec * 1e-6;
}

int main(){
  int n, blockSize, numBlocks;
  double a, b, h, integral;
  double* result;
  double* dev_result;

  printf("\nNumero de subintervalos: ");
  scanf("%d", &n);
  printf("\nLimite inicial: ");
  scanf("%lf", &a);
  printf("\nLimite final: ");
  scanf("%lf", &b);

  h = fabs(b - a) / n;

  blockSize = 256;
  numBlocks = (n + blockSize - 1) / blockSize;

  result = (double*)malloc(sizeof(double));
  hipMalloc((void**)&dev_result, sizeof(double));

  *result = 0.0;
  hipMemcpy(dev_result, result, sizeof(double), hipMemcpyHostToDevice);

  double start = getCurrentTime();
  integrate<<<numBlocks, blockSize>>>(a, b, n, h, dev_result);
  hipDeviceSynchronize();
  double end = getCurrentTime();

  hipMemcpy(result, dev_result, sizeof(double), hipMemcpyDeviceToHost);

  integral = (h / 2) * (f(a) + f(b) + *result);

  printf("\nLa integral es: %lf\n", integral);

  double elapsedTime = end - start;
  printf("Tiempo transcurrido: %.6f seconds\n", elapsedTime);

  int numThreads = numBlocks * blockSize;
  double sequentialTime = integral;
  double parallelTime = elapsedTime;
  double speedup = sequentialTime / parallelTime;
  double efficiency = speedup / numThreads;
  double scalability = sequentialTime / (parallelTime * numThreads);

  printf("Velocidad: %.2f\n", speedup);
  printf("Eficiencia: %.2f\n", efficiency);
  printf("Escalabilidad: %.2f\n", scalability);

  free(result);
  }